#include <hip/hip_runtime.h>

#include <stdio.h>

#include <iostream>
#include <stdlib.h>
#include <ctime>
#include <cstdlib>
#include <time.h>
#include <chrono>
#include <sys/time.h>
//#include <windows.h>

using namespace std;

#define ROW 20
#define COL 20
#define LOOPS 2
#define VERBOSE 1
#define NUM_THREAD 100

void PrintGrid(int* grid) {
    for (int i = 0; i < ROW; i++) {
        for (int j = 0; j < COL; j++) {
            printf("%d ", grid[i * COL + j]);
        }
        printf("\n");
    }
}

void SerialGameOfLife(int* grid) {
    // make copy of grid for output:
    int* result = (int*)malloc(ROW * COL * sizeof(int));
    // copy grid to result:
    for (long long i = 0; i < (ROW * COL); i++) {
        result[i] = grid[i];
    }

    for (long long i = 0; i < ROW; i++) {
        for (long long j = 0; j < COL; j++) {
            int cell = grid[i + j * ROW];
            // check if cell is at edge:
            if (i == 0) { // UPPER EDGE
                // check if cell is at corner:
                if (i == 0 && j == 0) { // UPPER LEFT
                    int numFriends = grid[i * COL + (j + 1)] + grid[(i + 1) * COL + (j + 1)] + grid[(i + 1) * COL + j];
                    if (cell == 1) {
                        if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                            result[i * COL + j] = 0;
                        }
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else if (i == 0 && j == COL - 1) { // UPPER RIGHT
                    int numFriends = grid[i * COL + (j - 1)] + grid[(i + 1) * COL + (j - 1)] + grid[(i + 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else {
                    int numFriends = grid[i * COL + (j - 1)] + grid[(i + 1) * COL + (j - 1)] + grid[(i + 1) * COL + j] + grid[(i + 1) * COL + (j + 1)] + grid[i * COL + (j + 1)];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                }
            } else if (i == ROW - 1) { // LOWER EDGE
                // check if cell is at corner:
                if (i == ROW - 1 && j == 0) { // LOWER LEFT
                    int numFriends = grid[i * COL + (j + 1)] + grid[(i - 1) * COL + (j + 1)] + grid[(i - 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else if (i == ROW - 1 && j == COL - 1) { // LOWER RIGHT
                    int numFriends = grid[i * COL + (j - 1)] + grid[(i - 1) * COL + (j - 1)] + grid[(i - 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else {
                    int numFriends = grid[i * COL + (j - 1)] + grid[(i - 1) * COL + (j - 1)] + grid[(i - 1) * COL + j] + grid[(i - 1) * COL + (j + 1)] + grid[i * COL + (j + 1)];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                }
            } else if (j == 0) { // LEFT EDGE
                // check if cell is at corner:
                if (i == 0 && j == 0) { // UPPER LEFT
                    int numFriends = grid[i * COL + (j + 1)] + grid[(i + 1) * COL + (j + 1)] + grid[(i + 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else if (i == ROW - 1 && j == 0) { // LOWER LEFT
                    int numFriends = grid[i * COL + (j + 1)] + grid[(i - 1) * COL + (j + 1)] + grid[(i - 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else {
                    int numFriends = grid[i * COL + (j + 1)] + grid[(i - 1) * COL + (j + 1)] + grid[(i - 1) * COL + j] + grid[(i + 1) * COL + (j + 1)] + grid[(i + 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                }
            } else if (j == COL - 1) { // RIGHT EDGE
                // check if cell is at corner:
                if (i == 0 && j == COL - 1) { // UPPER RIGHT
                    int numFriends = grid[i * COL + (j - 1)] + grid[(i + 1) * COL + (j - 1)] + grid[(i + 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else if (i == ROW - 1 && j == COL - 1) { // LOWER RIGHT
                    int numFriends = grid[i * COL + (j - 1)] + grid[(i - 1) * ROW + (j - 1)] + grid[(i - 1) * ROW + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                } else {
                    int numFriends = grid[i * COL + (j - 1)] + grid[(i - 1) * COL + (j - 1)] + grid[(i - 1) * COL + j] + grid[(i + 1) * COL + (j - 1)] + grid[(i + 1) * COL + j];
                    if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                        result[i * COL + j] = 0;
                    } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                        result[i * COL + j] = 1;
                    }
                }
            } else { // Generic cell
                int numFriends = grid[(i * COL) + j - 1] + grid[((i - 1) * COL) + j - 1] + grid[((i - 1) * COL) + j] + grid[((i - 1) * COL) + j + 1] + grid[((i) * COL) + j + 1] + grid[((i + 1) * COL) + j + 1] + grid[((i + 1) * COL) + j] + grid[((i + 1) * COL) + j - 1];
                if (numFriends < 2 || numFriends >= 4) { // cell dies from loneliness or overpopulation
                    result[i * COL + j] = 0;
                } else if (numFriends == 2 || numFriends == 3) { // cell becomes alive due to having 3/2 neighbors
                    result[i * COL + j] = 1;
                }
            }
        }
    }
    // copy result back into grid
    for (int i = 0; i < (ROW * COL); i++) {
        grid[i] = result[i];
    }
}

__global__ void GPUGameOfLife(int* grid) {
    __shared__ int s_arr[100];
    int neighbor = 0;

    int r = threadIdx.x % ROW;
    int c = threadIdx.x / ROW;
    int cell = grid[r + c * COL];
    int row = r - 1;
    int col = c - 1;

    s_arr[r + c * ROW] = grid[r + c * ROW];
    __syncthreads();
    for (int i = row; i < row + 3; i++) {
        for (int j = col; j < col + 3; j++) {
            if (i >= 0 && j >= 0) {
                if (i < ROW && j < COL) {
                    if (i != r || j != c) {
                        if (s_arr[i + j * ROW] == 1) {
                            neighbor += 1;
                        }
                    }
                }
            }
        }
    }
    if (cell == 1 && neighbor <= 1) {
        cell = 0;
    } else if (cell == 0 && neighbor == 2) {
        cell = 1;
    } else if (cell == 0 && neighbor == 3) {
        cell = 1;
    } else if (cell == 1 && neighbor >= 4) {
        cell = 0;
    }
    __syncthreads();
    grid[r + c * ROW] = cell;
}


//int main(int argc, char** argv) {

//    if (argc != 4) {
//        printf("Your command line is not valid. Please input like this.\n");
//        printf("Usage: %s numRow numCol numLoops\n", argv[0]);
//        return 1;
//    }
//
//    int numRow = atoi(argv[1]);
//    int numCol = atoi(argv[2]);
//    int numLoops = atoi(argv[3]);
int main() {
    //struct timeval serial_start, serial_end;
    //struct timeval gpu_start, gpu_end;
    srand(time(NULL));
    int *grid;
    int *d_grid;
    int *gpu_output;
    grid = (int *) malloc(ROW * COL * sizeof(int));
    gpu_output = (int *) malloc(ROW * COL * sizeof(int));
    hipMalloc((void **) &d_grid, ROW * COL * sizeof(int));

    // populate grid with random data
    for (long long i = 0; i < (ROW * COL); i++) {
        grid[i] = rand() % 2;
    }
    if (VERBOSE == 1) {
        printf("INITIAL STATE: \n");
        PrintGrid(grid);
    }

    // copy to gpu:
    hipMemcpy(d_grid, grid, ROW * COL * sizeof(int), hipMemcpyHostToDevice);

    //gettimeofday(&gpu_start, NULL);
    auto gpu_start_time = chrono::high_resolution_clock::now();
    for (int i = 0; i < LOOPS; i++) {
        GPUGameOfLife <<<1, NUM_THREAD>>>(d_grid);
    }

    hipMemcpy(gpu_output, d_grid, ROW * COL * sizeof(int), hipMemcpyDeviceToHost);
    //gettimeofday(&gpu_end, NULL);
    auto gpu_end_time = chrono::high_resolution_clock::now();
    if (VERBOSE) {
        printf("GPU: \n");
        printf("-------------\n");
        PrintGrid(gpu_output);
    }
    //gettimeofday(&serial_start, NULL);
    auto serial_start_time = chrono::high_resolution_clock::now();
    for (long long i = 0; i < LOOPS; i++) {
        SerialGameOfLife(grid);
        if (VERBOSE) {
            printf("CPU: \n");
            printf("-------------\n");
            PrintGrid(grid);
        }
    }
    //gettimeofday(&serial_end, NULL);
    auto serial_end_time = chrono::high_resolution_clock::now();

    if (VERBOSE == 1) {
        printf("FINAL STATE AFTER %d ITERATIONS: \n", LOOPS);
        PrintGrid(grid);
    }
    printf("ROWS: %d, COLS: %d, LOOPS: %d, NUM_THREADS: %d\n", ROW, COL, LOOPS, NUM_THREAD);
//    if (1 == 1) {
//        printf("endtime: %d, starttime: %d\n", serial_end.tv_sec, serial_start.tv_sec);
//    }
//    printf("SERIAL TIME (ms): %d\n", (serial_end.tv_sec - serial_start.tv_sec) * 1e6 + (serial_end.tv_usec - serial_start.tv_usec));
//    if (1 == 1) {
//        printf("endtime: %d, starttime: %d\n", gpu_end.tv_sec, gpu_start.tv_sec);
//    }
//    printf("GPU TIME (ms): %d\n", (gpu_end.tv_sec - gpu_start.tv_sec) * 1e6 + (gpu_end.tv_usec - gpu_start.tv_usec));

    auto gpu_time_diff = chrono::duration_cast<chrono::microseconds>(gpu_end_time - gpu_start_time);
    printf("GPU execution time: %d (ms)\n", gpu_time_diff.count());

    auto serial_time_diff = chrono::duration_cast<chrono::microseconds>(serial_end_time - serial_start_time);
    printf("Serial execution time: %d (ms)\n", serial_time_diff.count());

    hipFree(d_grid);
    free(gpu_output);
    free(grid);
    return 0;
}